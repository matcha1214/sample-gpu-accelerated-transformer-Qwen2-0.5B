#include "hip/hip_runtime.h"
#include "ArgMax.cuh"
#include <hip/hip_bf16.h>
#include "../ErrorCheck.h"

ArgMax::ArgMax(int32_t len) {
    size_t temp_size = len * sizeof(float) + sizeof(int32_t);
    temp_space = std::make_shared<CudaBuffer>(temp_size);
}

__global__ void argmax_kernel(__hip_bfloat16 *data, float *temp_values, int32_t *temp_indices, int32_t *result, int32_t len) {
    extern __shared__ float sdata[];
    int32_t *sindices = (int32_t*)&sdata[blockDim.x];
    
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    if (idx < len) {
        sdata[tid] = __bfloat162float(data[idx]);
        sindices[tid] = idx;
    } else {
        sdata[tid] = -INFINITY;
        sindices[tid] = -1;
    }
    
    __syncthreads();
    
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && idx + s < len) {
            if (sdata[tid + s] > sdata[tid] || 
                (sdata[tid + s] == sdata[tid] && sindices[tid + s] < sindices[tid])) {
                sdata[tid] = sdata[tid + s];
                sindices[tid] = sindices[tid + s];
            }
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        temp_values[blockIdx.x] = sdata[0];
        temp_indices[blockIdx.x] = sindices[0];
    }
}

__global__ void argmax_final_kernel(float *temp_values, int32_t *temp_indices, int32_t *result, int32_t num_blocks) {
    float max_val = -INFINITY;
    int32_t max_idx = -1;
    
    for (int i = 0; i < num_blocks; i++) {
        if (temp_values[i] > max_val || 
            (temp_values[i] == max_val && temp_indices[i] < max_idx)) {
            max_val = temp_values[i];
            max_idx = temp_indices[i];
        }
    }
    
    *result = max_idx;
}

int32_t *ArgMax::bf16_argmax(const std::shared_ptr<CudaBuffer> &bf16_data, hipStream_t stream) {
    __hip_bfloat16 *data = static_cast<__hip_bfloat16*>(bf16_data->data);
    int32_t len = bf16_data->size / sizeof(__hip_bfloat16);
    
    int32_t block_size = 256;
    int32_t grid_size = (len + block_size - 1) / block_size;
    
    float *temp_values = static_cast<float*>(temp_space->data);
    int32_t *temp_indices = reinterpret_cast<int32_t*>(temp_values + grid_size);
    int32_t *result = temp_indices + grid_size;
    
    size_t shared_mem_size = block_size * (sizeof(float) + sizeof(int32_t));
    argmax_kernel<<<grid_size, block_size, shared_mem_size, stream>>>(
        data, temp_values, temp_indices, result, len);
    
    argmax_final_kernel<<<1, 1, 0, stream>>>(temp_values, temp_indices, result, grid_size);
    
    return result;
}
